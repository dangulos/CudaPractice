#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       

// Max Threads per block in GeForce 210
#define TxB 1024

__global__
void reduction_kernel(const uchar4* const rgbaImage,
                       unsigned char* const outputImage,
                       int numRows, int numCols, int totalThreads)
{
  int aux = blockIdx.x * blockDim.x + threadIdx.x;
  int j = aux % 852;
  int i = (aux - j) / 852;
  int index = (j + i * 852) * 3;
  int x = j * (numCols/852.0);
  int y = i * (numRows/480.0);

  int indexAux = (x + y * numCols);
  uchar4 px = rgbaImage[indexAux]; // thread pixel to process
  greyImage[index + 2] = px.x; 
  greyImage[index + 1] = px.y; 
  greyImage[index] = px.z;
}

void reduction(uchar4 * const d_originalImage,
                  unsigned char* const d_resizeImage, size_t numRows, size_t numCols, int aBlockSize, int aGridSize)
{

  
  int totalThreads = aBlockSize * aGridSize;

  // const dim3 blockSize(aBlockSize, 1, 1);
  // const dim3 gridSize(aGridSize, 1, 1);
  long long int total_px = 852*480;  // total pixels
  long int grids_n = ceil(total_px / aBlockSize); // grids numer
  const dim3 blockSize(aBlockSize, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols, totalThreads);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}